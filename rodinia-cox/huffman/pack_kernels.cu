#include "hip/hip_runtime.h"
#ifndef _PACK_KERNELS_H_
#define _PACK_KERNELS_H_
#include "parameters.h"

__global__ static void pack2(unsigned int *srcData, unsigned int *cindex,
                             unsigned int *cindex2, unsigned int *dstData,
                             unsigned int original_num_block_elements) {
  unsigned int tid = blockDim.x * blockIdx.x + threadIdx.x;

  // source index
  unsigned int offset = tid * original_num_block_elements; // DPB,
  unsigned int bitsize = cindex[tid];

  // destination index
  unsigned int pos = cindex2[tid], dword = pos / 32, bit = pos % 32;

  unsigned int i, dw, tmp;
  dw = srcData[offset]; // load the first dword from srcData[]
  tmp = dw >> bit;      // cut off those bits that do not fit into the initial
                        // location in destData[]
  atomicOr(&dstData[dword], tmp); // fill up this initial location
  tmp = (bit == 0) ? 0 : (dw << 32 - bit);
  for (i = 1; i < bitsize / 32;
       i++) { // from now on, we have exclusive access to destData[]
    dw = srcData[offset + i]; // load next dword from srcData[]
    tmp |= dw >> bit;         // fill up tmp
    dstData[dword + i] = tmp; // write complete dword to destData[]
    tmp = (bit == 0) ? 0 : (dw << 32 - bit);
  }
  // exclusive access to dstData[] ends here
  // the remaining block can, or rather should be further optimized
  // write the remaining bits in tmp, UNLESS bit is 0 and bitsize is divisible
  // by 32, in this case do nothing
  if (bit != 0 || bitsize % 32 != 0)
    atomicOr(&dstData[dword + i], tmp);
  if (bitsize % 32 != 0) {
    dw = srcData[offset + i];
    atomicOr(&dstData[dword + i], dw >> bit);
    atomicOr(&dstData[dword + i + 1], (bit == 0) ? 0 : (dw << 32 - bit));
  }
}

#endif
