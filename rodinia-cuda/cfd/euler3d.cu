#include "hip/hip_runtime.h"
// Copyright 2009, Andrew Corrigan, acorriga@gmu.edu
// This code is from the AIAA-2009-4001 paper

//#include <cutil.h>
#include "hip/hip_runtime_api.h"
#include "helper_timer.h"
#include <iostream>
#include <fstream>

 
 
/*
 * Options 
 * 
 */ 
#define GAMMA 1.4f
#define iterations 2000
// #ifndef block_length
// 	#define block_length 192
// #endif



#define NDIM 3
#define NNB 4

#define RK 3	// 3rd order RK
#define ff_mach 1.2f
#define deg_angle_of_attack 0.0f

/*
 * not options
 */

#ifdef RD_WG_SIZE_0_0
	#define BLOCK_SIZE_0 RD_WG_SIZE_0_0
#elif defined(RD_WG_SIZE_0)
	#define BLOCK_SIZE_0 RD_WG_SIZE_0
#elif defined(RD_WG_SIZE)
	#define BLOCK_SIZE_0 RD_WG_SIZE
#else
	#define BLOCK_SIZE_0 192
#endif

#ifdef RD_WG_SIZE_1_0
	#define BLOCK_SIZE_1 RD_WG_SIZE_1_0
#elif defined(RD_WG_SIZE_1)
	#define BLOCK_SIZE_1 RD_WG_SIZE_1
#elif defined(RD_WG_SIZE)
	#define BLOCK_SIZE_1 RD_WG_SIZE
#else
	#define BLOCK_SIZE_1 192
#endif

#ifdef RD_WG_SIZE_2_0
	#define BLOCK_SIZE_2 RD_WG_SIZE_2_0
#elif defined(RD_WG_SIZE_1)
	#define BLOCK_SIZE_2 RD_WG_SIZE_2
#elif defined(RD_WG_SIZE)
	#define BLOCK_SIZE_2 RD_WG_SIZE
#else
	#define BLOCK_SIZE_2 192
#endif

#ifdef RD_WG_SIZE_3_0
	#define BLOCK_SIZE_3 RD_WG_SIZE_3_0
#elif defined(RD_WG_SIZE_3)
	#define BLOCK_SIZE_3 RD_WG_SIZE_3
#elif defined(RD_WG_SIZE)
	#define BLOCK_SIZE_3 RD_WG_SIZE
#else
	#define BLOCK_SIZE_3 192
#endif

#ifdef RD_WG_SIZE_4_0
	#define BLOCK_SIZE_4 RD_WG_SIZE_4_0
#elif defined(RD_WG_SIZE_4)
	#define BLOCK_SIZE_4 RD_WG_SIZE_4
#elif defined(RD_WG_SIZE)
	#define BLOCK_SIZE_4 RD_WG_SIZE
#else
	#define BLOCK_SIZE_4 192
#endif



// #if block_length > 128
// #warning "the kernels may fail too launch on some systems if the block length is too large"
// #endif


#define VAR_DENSITY 0
#define VAR_MOMENTUM  1
#define VAR_DENSITY_ENERGY (VAR_MOMENTUM+NDIM)
#define NVAR (VAR_DENSITY_ENERGY+1)


/*
 * Generic functions
 */

 #ifdef TIME_IT
 long long get_time() {
	 struct timeval tv;
	 gettimeofday(&tv, NULL);
	 return (tv.tv_sec * 1000000) + tv.tv_usec;
 }
 #endif


template <typename T>
#ifdef TIME_IT
T* alloc(int N, long long &time)
#else
T* alloc(int N)
#endif
{
	T* t;
    #ifdef TIME_IT
  	long long time1;
	long long time0 = get_time();
    #endif

	checkCudaErrors(hipMalloc((void**)&t, sizeof(T)*N));

	#ifdef TIME_IT
    hipDeviceSynchronize();
    time1 = get_time();
    time = time1-time0;
    #endif

    return t;
}

template <typename T>
#ifdef TIME_IT
long long dealloc(T* array)
#else
void dealloc(T* array)
#endif
{
	#ifdef TIME_IT
  	long long time1;
	long long time0 = get_time();
    #endif

	checkCudaErrors(hipFree((void*)array));

	#ifdef TIME_IT
    hipDeviceSynchronize();
    time1 = get_time();
    return time1-time0;
    #endif
}

template <typename T>
#ifdef TIME_IT
long long copy(T* dst, T* src, int N)
#else
void copy(T* dst, T* src, int N)
#endif
{
    #ifdef TIME_IT
  	long long time1;
	long long time0 = get_time();
    #endif
	checkCudaErrors(hipMemcpy((void*)dst, (void*)src, N*sizeof(T), hipMemcpyDeviceToDevice));
	#ifdef TIME_IT
    time1 = get_time();
    return time1-time0;
    #endif
}

template <typename T>
#ifdef TIME_IT
long long upload(T* dst, T* src, int N)
#else
void upload(T* dst, T* src, int N)
#endif
{
    #ifdef TIME_IT
  	long long time1;
	long long time0 = get_time();
    #endif
	checkCudaErrors(hipMemcpy((void*)dst, (void*)src, N*sizeof(T), hipMemcpyHostToDevice));
	#ifdef TIME_IT
    time1 = get_time();
    return time1-time0;
    #endif
}

template <typename T>
#ifdef TIME_IT
long long download(T* dst, T* src, int N)
#else
void download(T* dst, T* src, int N)
#endif
{
    #ifdef TIME_IT
  	long long time1;
	long long time0 = get_time();
    #endif
	checkCudaErrors(hipMemcpy((void*)dst, (void*)src, N*sizeof(T), hipMemcpyDeviceToHost));
	#ifdef TIME_IT
    time1 = get_time();
    return time1-time0;
    #endif
}

#ifdef TIME_IT
long long dump(float* variables, int nel, int nelr)
#else
void dump(float* variables, int nel, int nelr)
#endif
{
	float* h_variables = new float[nelr*NVAR];
    #ifdef TIME_IT
    long long time =
    #endif
	download(h_variables, variables, nelr*NVAR);

	{
		std::ofstream file("density");
		file << nel << " " << nelr << std::endl;
		for(int i = 0; i < nel; i++) file << h_variables[i + VAR_DENSITY*nelr] << std::endl;
	}


	{
		std::ofstream file("momentum");
		file << nel << " " << nelr << std::endl;
		for(int i = 0; i < nel; i++)
		{
			for(int j = 0; j != NDIM; j++)
				file << h_variables[i + (VAR_MOMENTUM+j)*nelr] << " ";
			file << std::endl;
		}
	}
	
	{
		std::ofstream file("density_energy");
		file << nel << " " << nelr << std::endl;
		for(int i = 0; i < nel; i++) file << h_variables[i + VAR_DENSITY_ENERGY*nelr] << std::endl;
	}
	delete[] h_variables;

	#ifdef TIME_IT
    return time;
    #endif
}

/*
 * Element-based Cell-centered FVM solver functions
 */
__constant__ float ff_variable[NVAR];
__constant__ float3 ff_flux_contribution_momentum_x[1];
__constant__ float3 ff_flux_contribution_momentum_y[1];
__constant__ float3 ff_flux_contribution_momentum_z[1];
__constant__ float3 ff_flux_contribution_density_energy[1];

__global__ void cuda_initialize_variables(int nelr, float* variables)
{
	const int i = (blockDim.x*blockIdx.x + threadIdx.x);
	for(int j = 0; j < NVAR; j++)
		variables[i + j*nelr] = ff_variable[j];
}
#ifdef TIME_IT
long long initialize_variables(int nelr, float* variables)
#else
void initialize_variables(int nelr, float* variables)
#endif
{
	dim3 Dg(nelr / BLOCK_SIZE_1), Db(BLOCK_SIZE_1);
	#ifdef TIME_IT
  	long long time1;
	long long time0 = get_time();
    #endif
	cuda_initialize_variables<<<Dg, Db>>>(nelr, variables);
	getLastCudaError("initialize_variables failed");
	#ifdef TIME_IT
	hipDeviceSynchronize();
	time1 = get_time();
    return time1-time0;
    #endif
}

__device__ __host__ inline void compute_flux_contribution(float& density, float3& momentum, float& density_energy, float& pressure, float3& velocity, float3& fc_momentum_x, float3& fc_momentum_y, float3& fc_momentum_z, float3& fc_density_energy)
{
	fc_momentum_x.x = velocity.x*momentum.x + pressure;
	fc_momentum_x.y = velocity.x*momentum.y;
	fc_momentum_x.z = velocity.x*momentum.z;
	
	
	fc_momentum_y.x = fc_momentum_x.y;
	fc_momentum_y.y = velocity.y*momentum.y + pressure;
	fc_momentum_y.z = velocity.y*momentum.z;

	fc_momentum_z.x = fc_momentum_x.z;
	fc_momentum_z.y = fc_momentum_y.z;
	fc_momentum_z.z = velocity.z*momentum.z + pressure;

	float de_p = density_energy+pressure;
	fc_density_energy.x = velocity.x*de_p;
	fc_density_energy.y = velocity.y*de_p;
	fc_density_energy.z = velocity.z*de_p;
}

__device__ inline void compute_velocity(float& density, float3& momentum, float3& velocity)
{
	velocity.x = momentum.x / density;
	velocity.y = momentum.y / density;
	velocity.z = momentum.z / density;
}
	
__device__ inline float compute_speed_sqd(float3& velocity)
{
	return velocity.x*velocity.x + velocity.y*velocity.y + velocity.z*velocity.z;
}

__device__ inline float compute_pressure(float& density, float& density_energy, float& speed_sqd)
{
	return (float(GAMMA)-float(1.0f))*(density_energy - float(0.5f)*density*speed_sqd);
}

__device__ inline float compute_speed_of_sound(float& density, float& pressure)
{
	return sqrtf(float(GAMMA)*pressure/density);
}

__global__ void cuda_compute_step_factor(int nelr, float* variables, float* areas, float* step_factors)
{
	const int i = (blockDim.x*blockIdx.x + threadIdx.x);

	float density = variables[i + VAR_DENSITY*nelr];
	float3 momentum;
	momentum.x = variables[i + (VAR_MOMENTUM+0)*nelr];
	momentum.y = variables[i + (VAR_MOMENTUM+1)*nelr];
	momentum.z = variables[i + (VAR_MOMENTUM+2)*nelr];
	
	float density_energy = variables[i + VAR_DENSITY_ENERGY*nelr];
	
	float3 velocity;       compute_velocity(density, momentum, velocity);
	float speed_sqd      = compute_speed_sqd(velocity);
	float pressure       = compute_pressure(density, density_energy, speed_sqd);
	float speed_of_sound = compute_speed_of_sound(density, pressure);

	// dt = float(0.5f) * sqrtf(areas[i]) /  (||v|| + c).... but when we do time stepping, this later would need to be divided by the area, so we just do it all at once
	step_factors[i] = float(0.5f) / (sqrtf(areas[i]) * (sqrtf(speed_sqd) + speed_of_sound));
}

#ifdef TIME_IT
long long compute_step_factor(int nelr, float* variables, float* areas, float* step_factors)
#else
void compute_step_factor(int nelr, float* variables, float* areas, float* step_factors)
#endif
{
	dim3 Dg(nelr / BLOCK_SIZE_2), Db(BLOCK_SIZE_2);
	#ifdef TIME_IT
	long long time1;
  	long long time0 = get_time();
  	#endif
	cuda_compute_step_factor<<<Dg, Db>>>(nelr, variables, areas, step_factors);		
	getLastCudaError("compute_step_factor failed");
    #ifdef TIME_IT
    hipDeviceSynchronize();
    time1 = get_time();
    return time1-time0;
    #endif
}

/*
 *
 *
*/
__global__ void cuda_compute_flux(int nelr, int* elements_surrounding_elements, float* normals, float* variables, float* fluxes)
{
	const float smoothing_coefficient = float(0.2f);
	const int i = (blockDim.x*blockIdx.x + threadIdx.x);
	
	int j, nb;
	float3 normal; float normal_len;
	float factor;
	
	float density_i = variables[i + VAR_DENSITY*nelr];
	float3 momentum_i;
	momentum_i.x = variables[i + (VAR_MOMENTUM+0)*nelr];
	momentum_i.y = variables[i + (VAR_MOMENTUM+1)*nelr];
	momentum_i.z = variables[i + (VAR_MOMENTUM+2)*nelr];

	float density_energy_i = variables[i + VAR_DENSITY_ENERGY*nelr];

	float3 velocity_i;             				compute_velocity(density_i, momentum_i, velocity_i);
	float speed_sqd_i                          = compute_speed_sqd(velocity_i);
	float speed_i                              = sqrtf(speed_sqd_i);
	float pressure_i                           = compute_pressure(density_i, density_energy_i, speed_sqd_i);
	float speed_of_sound_i                     = compute_speed_of_sound(density_i, pressure_i);
	float3 flux_contribution_i_momentum_x, flux_contribution_i_momentum_y, flux_contribution_i_momentum_z;
	float3 flux_contribution_i_density_energy;	
	compute_flux_contribution(density_i, momentum_i, density_energy_i, pressure_i, velocity_i, flux_contribution_i_momentum_x, flux_contribution_i_momentum_y, flux_contribution_i_momentum_z, flux_contribution_i_density_energy);
	
	float flux_i_density = float(0.0f);
	float3 flux_i_momentum;
	flux_i_momentum.x = float(0.0f);
	flux_i_momentum.y = float(0.0f);
	flux_i_momentum.z = float(0.0f);
	float flux_i_density_energy = float(0.0f);
		
	float3 velocity_nb;
	float density_nb, density_energy_nb;
	float3 momentum_nb;
	float3 flux_contribution_nb_momentum_x, flux_contribution_nb_momentum_y, flux_contribution_nb_momentum_z;
	float3 flux_contribution_nb_density_energy;	
	float speed_sqd_nb, speed_of_sound_nb, pressure_nb;
	
	#pragma unroll
	for(j = 0; j < NNB; j++)
	{
		nb = elements_surrounding_elements[i + j*nelr];
		normal.x = normals[i + (j + 0*NNB)*nelr];
		normal.y = normals[i + (j + 1*NNB)*nelr];
		normal.z = normals[i + (j + 2*NNB)*nelr];
		normal_len = sqrtf(normal.x*normal.x + normal.y*normal.y + normal.z*normal.z);
		
		if(nb >= 0) 	// a legitimate neighbor
		{
			density_nb = variables[nb + VAR_DENSITY*nelr];
			momentum_nb.x = variables[nb + (VAR_MOMENTUM+0)*nelr];
			momentum_nb.y = variables[nb + (VAR_MOMENTUM+1)*nelr];
			momentum_nb.z = variables[nb + (VAR_MOMENTUM+2)*nelr];
			density_energy_nb = variables[nb + VAR_DENSITY_ENERGY*nelr];
												compute_velocity(density_nb, momentum_nb, velocity_nb);
			speed_sqd_nb                      = compute_speed_sqd(velocity_nb);
			pressure_nb                       = compute_pressure(density_nb, density_energy_nb, speed_sqd_nb);
			speed_of_sound_nb                 = compute_speed_of_sound(density_nb, pressure_nb);
			                                    compute_flux_contribution(density_nb, momentum_nb, density_energy_nb, pressure_nb, velocity_nb, flux_contribution_nb_momentum_x, flux_contribution_nb_momentum_y, flux_contribution_nb_momentum_z, flux_contribution_nb_density_energy);
			
			// artificial viscosity
			factor = -normal_len*smoothing_coefficient*float(0.5f)*(speed_i + sqrtf(speed_sqd_nb) + speed_of_sound_i + speed_of_sound_nb);
			flux_i_density += factor*(density_i-density_nb);
			flux_i_density_energy += factor*(density_energy_i-density_energy_nb);
			flux_i_momentum.x += factor*(momentum_i.x-momentum_nb.x);
			flux_i_momentum.y += factor*(momentum_i.y-momentum_nb.y);
			flux_i_momentum.z += factor*(momentum_i.z-momentum_nb.z);

			// accumulate cell-centered fluxes
			factor = float(0.5f)*normal.x;
			flux_i_density += factor*(momentum_nb.x+momentum_i.x);
			flux_i_density_energy += factor*(flux_contribution_nb_density_energy.x+flux_contribution_i_density_energy.x);
			flux_i_momentum.x += factor*(flux_contribution_nb_momentum_x.x+flux_contribution_i_momentum_x.x);
			flux_i_momentum.y += factor*(flux_contribution_nb_momentum_y.x+flux_contribution_i_momentum_y.x);
			flux_i_momentum.z += factor*(flux_contribution_nb_momentum_z.x+flux_contribution_i_momentum_z.x);
			
			factor = float(0.5f)*normal.y;
			flux_i_density += factor*(momentum_nb.y+momentum_i.y);
			flux_i_density_energy += factor*(flux_contribution_nb_density_energy.y+flux_contribution_i_density_energy.y);
			flux_i_momentum.x += factor*(flux_contribution_nb_momentum_x.y+flux_contribution_i_momentum_x.y);
			flux_i_momentum.y += factor*(flux_contribution_nb_momentum_y.y+flux_contribution_i_momentum_y.y);
			flux_i_momentum.z += factor*(flux_contribution_nb_momentum_z.y+flux_contribution_i_momentum_z.y);
			
			factor = float(0.5f)*normal.z;
			flux_i_density += factor*(momentum_nb.z+momentum_i.z);
			flux_i_density_energy += factor*(flux_contribution_nb_density_energy.z+flux_contribution_i_density_energy.z);
			flux_i_momentum.x += factor*(flux_contribution_nb_momentum_x.z+flux_contribution_i_momentum_x.z);
			flux_i_momentum.y += factor*(flux_contribution_nb_momentum_y.z+flux_contribution_i_momentum_y.z);
			flux_i_momentum.z += factor*(flux_contribution_nb_momentum_z.z+flux_contribution_i_momentum_z.z);
		}
		else if(nb == -1)	// a wing boundary
		{
			flux_i_momentum.x += normal.x*pressure_i;
			flux_i_momentum.y += normal.y*pressure_i;
			flux_i_momentum.z += normal.z*pressure_i;
		}
		else if(nb == -2) // a far field boundary
		{
			factor = float(0.5f)*normal.x;
			flux_i_density += factor*(ff_variable[VAR_MOMENTUM+0]+momentum_i.x);
			flux_i_density_energy += factor*(ff_flux_contribution_density_energy[0].x+flux_contribution_i_density_energy.x);
			flux_i_momentum.x += factor*(ff_flux_contribution_momentum_x[0].x + flux_contribution_i_momentum_x.x);
			flux_i_momentum.y += factor*(ff_flux_contribution_momentum_y[0].x + flux_contribution_i_momentum_y.x);
			flux_i_momentum.z += factor*(ff_flux_contribution_momentum_z[0].x + flux_contribution_i_momentum_z.x);
			
			factor = float(0.5f)*normal.y;
			flux_i_density += factor*(ff_variable[VAR_MOMENTUM+1]+momentum_i.y);
			flux_i_density_energy += factor*(ff_flux_contribution_density_energy[0].y+flux_contribution_i_density_energy.y);
			flux_i_momentum.x += factor*(ff_flux_contribution_momentum_x[0].y + flux_contribution_i_momentum_x.y);
			flux_i_momentum.y += factor*(ff_flux_contribution_momentum_y[0].y + flux_contribution_i_momentum_y.y);
			flux_i_momentum.z += factor*(ff_flux_contribution_momentum_z[0].y + flux_contribution_i_momentum_z.y);

			factor = float(0.5f)*normal.z;
			flux_i_density += factor*(ff_variable[VAR_MOMENTUM+2]+momentum_i.z);
			flux_i_density_energy += factor*(ff_flux_contribution_density_energy[0].z+flux_contribution_i_density_energy.z);
			flux_i_momentum.x += factor*(ff_flux_contribution_momentum_x[0].z + flux_contribution_i_momentum_x.z);
			flux_i_momentum.y += factor*(ff_flux_contribution_momentum_y[0].z + flux_contribution_i_momentum_y.z);
			flux_i_momentum.z += factor*(ff_flux_contribution_momentum_z[0].z + flux_contribution_i_momentum_z.z);

		}
	}

	fluxes[i + VAR_DENSITY*nelr] = flux_i_density;
	fluxes[i + (VAR_MOMENTUM+0)*nelr] = flux_i_momentum.x;
	fluxes[i + (VAR_MOMENTUM+1)*nelr] = flux_i_momentum.y;
	fluxes[i + (VAR_MOMENTUM+2)*nelr] = flux_i_momentum.z;
	fluxes[i + VAR_DENSITY_ENERGY*nelr] = flux_i_density_energy;
}

#ifdef TIME_IT
long long compute_flux(int nelr, int* elements_surrounding_elements, float* normals, float* variables, float* fluxes)
#else
void compute_flux(int nelr, int* elements_surrounding_elements, float* normals, float* variables, float* fluxes)
#endif
{
	dim3 Dg(nelr / BLOCK_SIZE_3), Db(BLOCK_SIZE_3);
	#ifdef TIME_IT
  	long long time1;
	long long time0 = get_time();
    #endif
	cuda_compute_flux<<<Dg,Db>>>(nelr, elements_surrounding_elements, normals, variables, fluxes);
	getLastCudaError("compute_flux failed");

	#ifdef TIME_IT
    hipDeviceSynchronize();
    time1 = get_time();
    return time1-time0;
    #endif
}

__global__ void cuda_time_step(int j, int nelr, float* old_variables, float* variables, float* step_factors, float* fluxes)
{
	const int i = (blockDim.x*blockIdx.x + threadIdx.x);

	float factor = step_factors[i]/float(RK+1-j);

	variables[i + VAR_DENSITY*nelr] = old_variables[i + VAR_DENSITY*nelr] + factor*fluxes[i + VAR_DENSITY*nelr];
	variables[i + VAR_DENSITY_ENERGY*nelr] = old_variables[i + VAR_DENSITY_ENERGY*nelr] + factor*fluxes[i + VAR_DENSITY_ENERGY*nelr];
	variables[i + (VAR_MOMENTUM+0)*nelr] = old_variables[i + (VAR_MOMENTUM+0)*nelr] + factor*fluxes[i + (VAR_MOMENTUM+0)*nelr];
	variables[i + (VAR_MOMENTUM+1)*nelr] = old_variables[i + (VAR_MOMENTUM+1)*nelr] + factor*fluxes[i + (VAR_MOMENTUM+1)*nelr];	
	variables[i + (VAR_MOMENTUM+2)*nelr] = old_variables[i + (VAR_MOMENTUM+2)*nelr] + factor*fluxes[i + (VAR_MOMENTUM+2)*nelr];	
}

#ifdef TIME_IT
long long time_step(int j, int nelr, float* old_variables, float* variables, float* step_factors, float* fluxes)
#else
void time_step(int j, int nelr, float* old_variables, float* variables, float* step_factors, float* fluxes)
#endif
{
	dim3 Dg(nelr / BLOCK_SIZE_4), Db(BLOCK_SIZE_4);
	#ifdef TIME_IT
  	long long time1;
	long long time0 = get_time();
    #endif
	cuda_time_step<<<Dg,Db>>>(j, nelr, old_variables, variables, step_factors, fluxes);
	getLastCudaError("update failed");
	#ifdef TIME_IT
    hipDeviceSynchronize();
    time1 = get_time();
    return time1-time0;
    #endif
}

/*
 * Main function
 */
int main(int argc, char** argv)
{
	#ifdef TIME_IT
    long long initTime = 0;
    long long alocTime = 0;
    long long cpInTime = 0;
    long long kernTime = 0;
    long long cpOtTime = 0;
    long long freeTime = 0;
    long long auxTime1 = 0;
    long long auxTime2 = 0;
    #endif

  printf("WG size of kernel:initialize = %d, WG size of kernel:compute_step_factor = %d, WG size of kernel:compute_flux = %d, WG size of kernel:time_step = %d\n", BLOCK_SIZE_1, BLOCK_SIZE_2, BLOCK_SIZE_3, BLOCK_SIZE_4);

	if (argc < 2)
	{
		std::cout << "specify data file name" << std::endl;
		return 0;
	}
	const char* data_file_name = argv[1];
	
	hipDeviceProp_t prop;
	int dev;
	
	#ifdef TIME_IT
    auxTime1 = get_time();
	#endif
	checkCudaErrors(hipSetDevice(0));
	checkCudaErrors(hipGetDevice(&dev));
	checkCudaErrors(hipGetDeviceProperties(&prop, dev));
	#ifdef TIME_IT
	auxTime2 = get_time();
    initTime = auxTime2-auxTime1;
	#endif
	
	printf("Name:                     %s\n", prop.name);

	// set far field conditions and load them into constant memory on the gpu
	{
		float h_ff_variable[NVAR];
		const float angle_of_attack = float(3.1415926535897931 / 180.0f) * float(deg_angle_of_attack);
		
		h_ff_variable[VAR_DENSITY] = float(1.4);
		
		float ff_pressure = float(1.0f);
		float ff_speed_of_sound = sqrt(GAMMA*ff_pressure / h_ff_variable[VAR_DENSITY]);
		float ff_speed = float(ff_mach)*ff_speed_of_sound;
		
		float3 ff_velocity;
		ff_velocity.x = ff_speed*float(cos((float)angle_of_attack));
		ff_velocity.y = ff_speed*float(sin((float)angle_of_attack));
		ff_velocity.z = 0.0f;
		
		h_ff_variable[VAR_MOMENTUM+0] = h_ff_variable[VAR_DENSITY] * ff_velocity.x;
		h_ff_variable[VAR_MOMENTUM+1] = h_ff_variable[VAR_DENSITY] * ff_velocity.y;
		h_ff_variable[VAR_MOMENTUM+2] = h_ff_variable[VAR_DENSITY] * ff_velocity.z;
				
		h_ff_variable[VAR_DENSITY_ENERGY] = h_ff_variable[VAR_DENSITY]*(float(0.5f)*(ff_speed*ff_speed)) + (ff_pressure / float(GAMMA-1.0f));

		float3 h_ff_momentum;
		h_ff_momentum.x = *(h_ff_variable+VAR_MOMENTUM+0);
		h_ff_momentum.y = *(h_ff_variable+VAR_MOMENTUM+1);
		h_ff_momentum.z = *(h_ff_variable+VAR_MOMENTUM+2);
		float3 h_ff_flux_contribution_momentum_x;
		float3 h_ff_flux_contribution_momentum_y;
		float3 h_ff_flux_contribution_momentum_z;
		float3 h_ff_flux_contribution_density_energy;
		compute_flux_contribution(h_ff_variable[VAR_DENSITY], h_ff_momentum, h_ff_variable[VAR_DENSITY_ENERGY], ff_pressure, ff_velocity, h_ff_flux_contribution_momentum_x, h_ff_flux_contribution_momentum_y, h_ff_flux_contribution_momentum_z, h_ff_flux_contribution_density_energy);

		// copy far field conditions to the gpu
		#ifdef TIME_IT
        auxTime1 = get_time();
		#endif
		checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(ff_variable),          h_ff_variable,          NVAR*sizeof(float)) );
		checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(ff_flux_contribution_momentum_x), &h_ff_flux_contribution_momentum_x, sizeof(float3)) );
		checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(ff_flux_contribution_momentum_y), &h_ff_flux_contribution_momentum_y, sizeof(float3)) );
		checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(ff_flux_contribution_momentum_z), &h_ff_flux_contribution_momentum_z, sizeof(float3)) );
		
		checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(ff_flux_contribution_density_energy), &h_ff_flux_contribution_density_energy, sizeof(float3)) );
		#ifdef TIME_IT
		auxTime2 = get_time();
        cpInTime += auxTime2-auxTime1;
		#endif	
	}
	int nel;
	int nelr;
	
	// read in domain geometry
	float* areas;
	int* elements_surrounding_elements;
	float* normals;
	{
		std::ifstream file(data_file_name);
	
		file >> nel;
		nelr = BLOCK_SIZE_0*((nel / BLOCK_SIZE_0 )+ std::min(1, nel % BLOCK_SIZE_0));

		float* h_areas = new float[nelr];
		int* h_elements_surrounding_elements = new int[nelr*NNB];
		float* h_normals = new float[nelr*NDIM*NNB];

				
		// read in data
		for(int i = 0; i < nel; i++)
		{
			file >> h_areas[i];
			for(int j = 0; j < NNB; j++)
			{
				file >> h_elements_surrounding_elements[i + j*nelr];
				if(h_elements_surrounding_elements[i+j*nelr] < 0) h_elements_surrounding_elements[i+j*nelr] = -1;
				h_elements_surrounding_elements[i + j*nelr]--; //it's coming in with Fortran numbering				
				
				for(int k = 0; k < NDIM; k++)
				{
					file >> h_normals[i + (j + k*NNB)*nelr];
					h_normals[i + (j + k*NNB)*nelr] = -h_normals[i + (j + k*NNB)*nelr];
				}
			}
		}
		
		// fill in remaining data
		int last = nel-1;
		for(int i = nel; i < nelr; i++)
		{
			h_areas[i] = h_areas[last];
			for(int j = 0; j < NNB; j++)
			{
				// duplicate the last element
				h_elements_surrounding_elements[i + j*nelr] = h_elements_surrounding_elements[last + j*nelr];	
				for(int k = 0; k < NDIM; k++) h_normals[last + (j + k*NNB)*nelr] = h_normals[last + (j + k*NNB)*nelr];
			}
		}
		
		#ifdef TIME_IT
		areas = alloc<float>(nelr, auxTime1);
        alocTime += auxTime1;
        
        cpInTime += upload<float>(areas, h_areas, nelr);

        elements_surrounding_elements = alloc<int>(nelr*NNB, auxTime1);
        alocTime += auxTime1;
		cpInTime += upload<int>(elements_surrounding_elements, h_elements_surrounding_elements, nelr*NNB);

		normals = alloc<float>(nelr*NDIM*NNB, auxTime1);
        alocTime += auxTime1;
		cpInTime += upload<float>(normals, h_normals, nelr*NDIM*NNB);
        #else
		areas = alloc<float>(nelr);
		upload<float>(areas, h_areas, nelr);

		elements_surrounding_elements = alloc<int>(nelr*NNB);
		upload<int>(elements_surrounding_elements, h_elements_surrounding_elements, nelr*NNB);

		normals = alloc<float>(nelr*NDIM*NNB);
		upload<float>(normals, h_normals, nelr*NDIM*NNB);
		#endif

		delete[] h_areas;
		delete[] h_elements_surrounding_elements;
		delete[] h_normals;
	}

	// Create arrays and set initial conditions

	#ifdef TIME_IT
    float* variables = alloc<float>(nelr*NVAR, auxTime1);
    alocTime += auxTime1;

    kernTime += initialize_variables(nelr, variables);

    float* old_variables = alloc<float>(nelr*NVAR, auxTime1); 
    alocTime += auxTime1;

	float* fluxes = alloc<float>(nelr*NVAR, auxTime1);
    alocTime += auxTime1;

	float* step_factors = alloc<float>(nelr, auxTime1);
    alocTime += auxTime1;

    kernTime += initialize_variables(nelr, old_variables);
    kernTime += initialize_variables(nelr, fluxes);
	hipMemset( (void*) step_factors, 0, sizeof(float)*nelr );
	#else

	float* variables = alloc<float>(nelr*NVAR);
	initialize_variables(nelr, variables);

	float* old_variables = alloc<float>(nelr*NVAR);   	
	float* fluxes = alloc<float>(nelr*NVAR);
	float* step_factors = alloc<float>(nelr); 

	// make sure all memory is floatly allocated before we start timing
	initialize_variables(nelr, old_variables);
	initialize_variables(nelr, fluxes);

	hipMemset( (void*) step_factors, 0, sizeof(float)*nelr );
	#endif
	// make sure CUDA isn't still doing something before we start timing
	hipDeviceSynchronize();

	// these need to be computed the first time in order to compute time step
	std::cout << "Starting..." << std::endl;

	StopWatchInterface *timer = 0;
	  //	unsigned int timer = 0;

	// CUT_SAFE_CALL( cutCreateTimer( &timer));
	// CUT_SAFE_CALL( cutStartTimer( timer));
	sdkCreateTimer(&timer); 
	sdkStartTimer(&timer); 
	// Begin iterations
	for(int i = 0; i < iterations; i++)
	{
		#ifdef TIME_IT
        copy<float>(old_variables, variables, nelr*NVAR);
        kernTime += compute_step_factor(nelr, variables, areas, step_factors);

        for(int j = 0; j < RK; j++)
		{
			kernTime += compute_flux(nelr, elements_surrounding_elements, normals, variables, fluxes);
			kernTime += time_step(j, nelr, old_variables, variables, step_factors, fluxes);
		}
        #else
		copy<float>(old_variables, variables, nelr*NVAR);
		
		// for the first iteration we compute the time step
		compute_step_factor(nelr, variables, areas, step_factors);
		getLastCudaError("compute_step_factor failed");
		
		for(int j = 0; j < RK; j++)
		{
			compute_flux(nelr, elements_surrounding_elements, normals, variables, fluxes);
			getLastCudaError("compute_flux failed");			
			time_step(j, nelr, old_variables, variables, step_factors, fluxes);
			getLastCudaError("time_step failed");			
		}
		#endif
	}

	hipDeviceSynchronize();
	//	CUT_SAFE_CALL( cutStopTimer(timer) );  
	sdkStopTimer(&timer); 

	std::cout  << (sdkGetAverageTimerValue(&timer)/1000.0)  / iterations << " seconds per iteration" << std::endl;

	std::cout << "Saving solution..." << std::endl;
	#ifdef TIME_IT
	cpOtTime += 
    #endif
	dump(variables, nel, nelr);
	std::cout << "Saved solution..." << std::endl;

	
	std::cout << "Cleaning up..." << std::endl;
	#ifdef TIME_IT
	freeTime += 
    #endif
	dealloc<float>(areas);
	#ifdef TIME_IT
	freeTime += 
    #endif
	dealloc<int>(elements_surrounding_elements);
	#ifdef TIME_IT
	freeTime += 
    #endif
	dealloc<float>(normals);
	
	#ifdef TIME_IT
	freeTime += 
    #endif
	dealloc<float>(variables);
	#ifdef TIME_IT
	freeTime += 
    #endif
	dealloc<float>(old_variables);
	#ifdef TIME_IT
	freeTime += 
    #endif
	dealloc<float>(fluxes);
	#ifdef TIME_IT
	freeTime += 
    #endif
	dealloc<float>(step_factors);

	std::cout << "Done..." << std::endl;

	#ifdef TIME_IT
    long long totalTime = initTime + alocTime + cpInTime + kernTime + cpOtTime + freeTime;
	printf("Time spent in different stages of GPU_CUDA KERNEL:\n");

	printf("%15.12f s, %15.12f % : GPU: SET DEVICE / DRIVER INIT\n",	(float) initTime / 1000000, (float) initTime / (float) totalTime * 100);
	printf("%15.12f s, %15.12f % : GPU MEM: ALO\n", 					(float) alocTime / 1000000, (float) alocTime / (float) totalTime * 100);
	printf("%15.12f s, %15.12f % : GPU MEM: COPY IN\n",					(float) cpInTime / 1000000, (float) cpInTime / (float) totalTime * 100);

	printf("%15.12f s, %15.12f % : GPU: KERNEL\n",						(float) kernTime / 1000000, (float) kernTime / (float) totalTime * 100);

	printf("%15.12f s, %15.12f % : GPU MEM: COPY OUT\n",				(float) cpOtTime / 1000000, (float) cpOtTime / (float) totalTime * 100);
	printf("%15.12f s, %15.12f % : GPU MEM: FRE\n", 					(float) freeTime / 1000000, (float) freeTime / (float) totalTime * 100);

	printf("Total time:\n");
	printf("%.12f s\n", 												(float) totalTime / 1000000);
	#endif

	return 0;
}
