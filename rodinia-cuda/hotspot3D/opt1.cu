#include "hip/hip_runtime.h"
long long get_time() {
	struct timeval tv;
	gettimeofday(&tv, NULL);
	return (tv.tv_sec * 1000000) + tv.tv_usec;
}

__global__ void hotspotOpt1(float *p, float* tIn, float *tOut, float sdc,
        int nx, int ny, int nz,
        float ce, float cw, 
        float cn, float cs,
        float ct, float cb, 
        float cc) 
{
    float amb_temp = 80.0;

    int i = blockDim.x * blockIdx.x + threadIdx.x;  
    int j = blockDim.y * blockIdx.y + threadIdx.y;

    int c = i + j * nx;
    int xy = nx * ny;

    int W = (i == 0)        ? c : c - 1;
    int E = (i == nx-1)     ? c : c + 1;
    int N = (j == 0)        ? c : c - nx;
    int S = (j == ny-1)     ? c : c + nx;

    float temp1, temp2, temp3;
    temp1 = temp2 = tIn[c];
    temp3 = tIn[c+xy];
    tOut[c] = cc * temp2 + cw * tIn[W] + ce * tIn[E] + cs * tIn[S]
        + cn * tIn[N] + cb * temp1 + ct * temp3 + sdc * p[c] + ct * amb_temp;
    c += xy;
    W += xy;
    E += xy;
    N += xy;
    S += xy;

    for (int k = 1; k < nz-1; ++k) {
        temp1 = temp2;
        temp2 = temp3;
        temp3 = tIn[c+xy];
        tOut[c] = cc * temp2 + cw * tIn[W] + ce * tIn[E] + cs * tIn[S]
            + cn * tIn[N] + cb * temp1 + ct * temp3 + sdc * p[c] + ct * amb_temp;
        c += xy;
        W += xy;
        E += xy;
        N += xy;
        S += xy;
    }
    temp1 = temp2;
    temp2 = temp3;
    tOut[c] = cc * temp2 + cw * tIn[W] + ce * tIn[E] + cs * tIn[S]
        + cn * tIn[N] + cb * temp1 + ct * temp3 + sdc * p[c] + ct * amb_temp;
    return;
}

void hotspot_opt1(float *p, float *tIn, float *tOut,
        int nx, int ny, int nz,
        float Cap, 
        float Rx, float Ry, float Rz, 
        float dt, int numiter) 
{
    #ifdef TIME_IT
    long long initTime;
    long long alocTime = 0;
    long long cpinTime = 0;
    long long kernTime = 0;
    long long cpouTime = 0;
    long long freeTime = 0;
    long long aux1Time;
    long long aux2Time;
    #endif

    #ifdef TIME_IT
    aux1Time = get_time();
    hipDeviceSynchronize();
    aux2Time = get_time();
    initTime = aux2Time-aux1Time;
    #endif
  
    float ce, cw, cn, cs, ct, cb, cc;
    float stepDivCap = dt / Cap;
    ce = cw =stepDivCap/ Rx;
    cn = cs =stepDivCap/ Ry;
    ct = cb =stepDivCap/ Rz;

    cc = 1.0 - (2.0*ce + 2.0*cn + 3.0*ct);

    size_t s = sizeof(float) * nx * ny * nz;  
    float  *tIn_d, *tOut_d, *p_d;

    #ifdef TIME_IT
    aux1Time = get_time();
    #endif
    hipMalloc((void**)&p_d,s);
    hipMalloc((void**)&tIn_d,s);
    hipMalloc((void**)&tOut_d,s);
    #ifdef TIME_IT
    aux2Time = get_time();
    alocTime += aux2Time-aux1Time;
    aux1Time = get_time();
    #endif
    hipMemcpy(tIn_d, tIn, s, hipMemcpyHostToDevice);
    hipMemcpy(p_d, p, s, hipMemcpyHostToDevice);
    #ifdef TIME_IT
    aux2Time = get_time();
    cpinTime += aux2Time-aux1Time;
    #endif

    hipFuncSetCacheConfig(reinterpret_cast<const void*>(hotspotOpt1), hipFuncCachePreferL1);

    dim3 block_dim(64, 4, 1);
    dim3 grid_dim(nx / 64, ny / 4, 1);

    long long start = get_time();
    for (int i = 0; i < numiter; ++i) {
        hotspotOpt1<<<grid_dim, block_dim>>>
            (p_d, tIn_d, tOut_d, stepDivCap, nx, ny, nz, ce, cw, cn, cs, ct, cb, cc);
        float *t = tIn_d;
        tIn_d = tOut_d;
        tOut_d = t;
    }
    hipDeviceSynchronize();
    long long stop = get_time();
    #ifdef TIME_IT
    kernTime = stop-start;
    #endif
    float time = (float)((stop - start)/(1000.0 * 1000.0));
    printf("Time: %.3f (s)\n",time);
    #ifdef TIME_IT
    aux1Time = get_time();
    #endif
    hipMemcpy(tOut, tOut_d, s, hipMemcpyDeviceToHost);
    #ifdef TIME_IT
    aux2Time = get_time();
    cpouTime += aux2Time-aux1Time;
    aux1Time = get_time();
    #endif
    hipFree(p_d);
    hipFree(tIn_d);
    hipFree(tOut_d);
    #ifdef TIME_IT
    aux2Time = get_time();
    freeTime += aux2Time-aux1Time;
    #endif

    #ifdef TIME_IT
    long long totalTime = initTime + alocTime + cpinTime + kernTime + cpouTime + freeTime;
	printf("Time spent in different stages of GPU_CUDA KERNEL:\n");

	printf("%15.12f s, %15.12f % : GPU: SET DEVICE / DRIVER INIT\n",	(float) initTime / 1000000, (float) initTime / (float) totalTime * 100);
	printf("%15.12f s, %15.12f % : GPU MEM: ALO\n", 					(float) alocTime / 1000000, (float) alocTime / (float) totalTime * 100);
	printf("%15.12f s, %15.12f % : GPU MEM: COPY IN\n",					(float) cpinTime / 1000000, (float) cpinTime / (float) totalTime * 100);

	printf("%15.12f s, %15.12f % : GPU: KERNEL\n",						(float) kernTime / 1000000, (float) kernTime / (float) totalTime * 100);

	printf("%15.12f s, %15.12f % : GPU MEM: COPY OUT\n",				(float) cpouTime / 1000000, (float) cpouTime / (float) totalTime * 100);
	printf("%15.12f s, %15.12f % : GPU MEM: FRE\n", 					(float) freeTime / 1000000, (float) freeTime / (float) totalTime * 100);

	printf("Total time:\n");
	printf("%.12f s\n", 												(float) totalTime / 1000000);
	#endif
    
    return;
}

