/*Crown Copyright 2012 AWE.
 *
 * This file is part of CloverLeaf.
 *
 * CloverLeaf is free software: you can redistribute it and/or modify it under
 * the terms of the GNU General Public License as published by the
 * Free Software Foundation, either version 3 of the License, or (at your option)
 * any later version.
 *
 * CloverLeaf is distributed in the hope that it will be useful, but
 * WITHOUT ANY WARRANTY; without even the implied warranty of MERCHANTABILITY or
 * FITNESS FOR A PARTICULAR PURPOSE. See the GNU General Public License for more
 * details.
 *
 * You should have received a copy of the GNU General Public License along with
 * CloverLeaf. If not, see http://www.gnu.org/licenses/.
 */

/*
 *  @brief CUDA field summary kernel
 *  @author Michael Boulton NVIDIA Corporation
 *  @details The total mass, internal energy, kinetic energy and volume weighted
 *  pressure for the chunk is calculated.
 */

#include "cuda_common.hpp"
#include "kernel_files/field_summary_kernel.cuknl"

extern "C" void field_summary_kernel_cuda_(double *vol, double *mass, double *ie, double *ke, double *press)
{
    cuda_chunk.field_summary_kernel(vol, mass, ie, ke, press);
}

void CloverleafCudaChunk::field_summary_kernel(double *vol, double *mass, double *ie, double *ke, double *press)
{
    CUDALAUNCH(device_field_summary_kernel_cuda, volume, density0,
               energy0, pressure, xvel0, yvel0,
               reduce_buf_1, reduce_buf_2, reduce_buf_3,
               reduce_buf_4, reduce_buf_5);
    double *tmp = (double *)malloc(sizeof(double) * num_blocks);
    hipMemcpy(tmp, reduce_buf_1, sizeof(double) * num_blocks, hipMemcpyDeviceToHost);
    vol[0] = 0;
    for (int i = 0; i < num_blocks; i++)
        vol[0] += tmp[i];
    // ReduceToHost<double>::sum(reduce_buf_1, vol, num_blocks);
    hipMemcpy(tmp, reduce_buf_2, sizeof(double) * num_blocks, hipMemcpyDeviceToHost);
    mass[0] = 0;
    for (int i = 0; i < num_blocks; i++) {
        mass[0] += tmp[i];
    }

    // ReduceToHost<double>::sum(reduce_buf_2, mass, num_blocks);
    hipMemcpy(tmp, reduce_buf_3, sizeof(double) * num_blocks, hipMemcpyDeviceToHost);
    ie[0] = 0;
    for (int i = 0; i < num_blocks; i++)
        ie[0] += tmp[i];
    // ReduceToHost<double>::sum(reduce_buf_3, ie, num_blocks);
    hipMemcpy(tmp, reduce_buf_4, sizeof(double) * num_blocks, hipMemcpyDeviceToHost);
    ke[0] = 0;
    for (int i = 0; i < num_blocks; i++)
        ke[0] += tmp[i];
    // ReduceToHost<double>::sum(reduce_buf_4, ke, num_blocks);
    hipMemcpy(tmp, reduce_buf_5, sizeof(double) * num_blocks, hipMemcpyDeviceToHost);
    press[0] = 0;
    for (int i = 0; i < num_blocks; i++)
        press[0] += tmp[i];
    // ReduceToHost<double>::sum(reduce_buf_5, press, num_blocks);
}
